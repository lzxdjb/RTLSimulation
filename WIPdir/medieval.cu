
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <algorithm>
#include <vector>





struct Network {
    // Handled by srcKernel
    int sourceCnt;              // number of source nodes
    int* sourceIDs;             // sourceIDs[i] = ID of the i-th source node
    int* sourceBits;            // decimal-formatted source vals for k-th cycle are found through indices: k*sourceCnt -> (k+1)*sourceCnt-1

    // Handled by genKernel
    int nodeCnt;                // number of nodes
    int edgeCnt;                // number of edges
    int colCnt;                 // number of columns
    int lutCnt;                 // number of LUTs
    int regCnt;                 // number of registers
    int* offsets;               // offsets[i] = prefix sum of #prevNodes from node 0 to i-1, inclusive
    int* csr;                   // i-th node's prevNode IDs are found:     csr[offs[i]] -> csr[offs[i+1]-1]
    int* typeIDs;               // typeIDs[i] = n where node with ID i is the n-th input, LUT, or register, 0-indexed (to get INP/LUT/REG-ID given nodeID)
    int* regIndicator;          // regIndicator[i] = 1 if node @ ID i is a register, 0 otherwise.
    int* regBuffer;             // regBuffer[i] = regID i register's oldGen val before being touched by genKernel, newGen val after being touched.
    int* lutOffsets;            // lutOffs[i] = prefix sum of #ints-needed-to-represent-LUT-control-bits from LUT 0 to i-1, inclusive (offsets for the lutBits-CSR).
    unsigned int* lutBits;      // i-th LUT's lutbits (decimal-formatted) are found:         lutBits[lutOffsets[i]] -> lutBits[lutOffsets[i+1]-1]
    int* outputBits;            // decimal-formatted output vals for k-th cycle are found through indices: k*nodeCnt -> (k+1)*nodeCnt-1
    int* colIdxs;               // colIdxs[i] = prefix sum of column-nodecounts from col 0 to i-1, inclusive
    int* colIDs;                // i-th column's nodeIDs are found:        colIDs[colIdxs[i]] -> colIDs[colIdxs[i+1]-1]

    Network(int srcCnt, int* srcIDs, int* srcBts, int ndCnt, int edgCnt, int colCnt, int lutCnt, int regCnt, int* offs, int* csr,
            int* tpIDs, int* regInd, int* regBuf, int* lutOffs, unsigned int* lutBts, int* outBts, int* colIdxs, int* colIDs)
       : sourceCnt(srcCnt),
         sourceIDs(srcIDs),
         sourceBits(srcBts),
         nodeCnt(ndCnt),
         edgeCnt(edgCnt),
         colCnt(colCnt),
         lutCnt(lutCnt),
         regCnt(regCnt),
         offsets(offs),
         csr(csr),
         typeIDs(tpIDs),
         regIndicator(regInd),
         regBuffer(regBuf),
         lutOffsets(lutOffs),
         lutBits(lutBts),
         outputBits(outBts),
         colIdxs(colIdxs),
         colIDs(colIDs) {}
};


int comparator(const void *a, const void *b) {
    return ((const int (*)[2])a)[0][0] - ((const int (*)[2])b)[0][0];
}

// This csr-reversal function WILL NOT preserve the order of dependencies (matters to LUT computation)
void gen_outgoing_csr(Network ntwk, int* out_offs, int* out_csr) {
    // Form tuples of all (srcID, dstID) connexions from the graph
    int tupleCnt = 0;
    int tuples[ntwk.edgeCnt][2];         // k-th tuple located at: (tuples[k][0], tuples[k][1]])
    int* outDegrees = new int[ntwk.nodeCnt];        // outDegrees[nodeID] = out-degree of corresponding node

    for (int dstID=0; dstID<=ntwk.nodeCnt; dstID++) {
        int numDependencies = ntwk.offsets[dstID+1]-ntwk.offsets[dstID];
        for (int i=0; i<numDependencies; i++) {
            int srcID = ntwk.csr[ntwk.offsets[dstID]+i];
            tuples[tupleCnt][0] = srcID;
            tuples[tupleCnt][1] = dstID;
            outDegrees[srcID]++;
            tupleCnt++;
        }
    }

    // Sort tuples by srcID, in ascending order
    qsort((void*)tuples, ntwk.edgeCnt, sizeof(int*), comparator);

    // Configure offsets for outgoing-csr
    out_offs[0] = 0;
    int appendCntr = 0;
    for (int source=0; source<ntwk.nodeCnt; source++) {
        out_offs[source+1] = out_offs[source] + outDegrees[source];
        for (int i=0; i<outDegrees[source]; i++) {
            out_csr[out_offs[source]+i] = tuples[appendCntr][1];
            appendCntr++;
        }
    }

    free(outDegrees);
}

// Columnizes a network. Assumes that all fields are set except for colIdxs and colIDs
int columnize(Network ntwk) {
    int colCounter = 0;
    int nCnt = ntwk.nodeCnt;
    int sCnt = ntwk.sourceCnt;
    int curSourcesLength = sCnt;
    int* inDegrees = new int[nCnt];
    int* curSources = new int[nCnt];
    int* frontier = new int[nCnt];
    int* frwrd_offsets = new int[ntwk.nodeCnt+1];
    int* frwrd_csr = new int[ntwk.edgeCnt];

    // Generate forward-edge (outgoing) graph representation
    gen_outgoing_csr(ntwk, frwrd_offsets, frwrd_csr);

    // Populate inDegrees
    for (int k=0; k<nCnt; k++) {
        inDegrees[k] = ntwk.offsets[k+1]-ntwk.offsets[k];
    }

    // Populate curSources with source IDs to begin with
    memcpy(curSources, ntwk.sourceIDs, sCnt*sizeof(int));
    int curColIDsLength = 0;

    // Loop until all nodes have been placed into a column
    while (curSourcesLength != 0) {
        // Initialize next entry in colIdxs to continue the prefix sum
        ntwk.colIdxs[colCounter+1] = ntwk.colIdxs[colCounter];
        int frontierLength = 0;

        for (int i=0; i<curSourcesLength; i++) {
            // Flush each curSource to a column and decrement its inDegree so it isn't picked up again
            int nodeID = curSources[i];
            ntwk.colIDs[curColIDsLength] = nodeID;
            curColIDsLength++;
            inDegrees[nodeID]--;

            // Bump the current column's offset
            ntwk.colIdxs[colCounter+1]++;

            // Decrement inDegree count for each node outgoing from a curSource
            int outDeg = frwrd_offsets[nodeID+1]-frwrd_offsets[nodeID];
            for (int k=0; k<outDeg; k++) {
                inDegrees[frwrd_csr[frwrd_offsets[nodeID]+k]]--;
            }
        }

        // Append all nodes that have exactly 0 inDegree to the frontier
        for (int j=0; j<nCnt; j++) {
            if (inDegrees[j] == 0) {
                frontier[frontierLength] = j;
                frontierLength++;
            }
        }

        // Flush frontier to curSources
        // I think this can all be optimized by actually dual-buffering here. Too much memcpy...
        memset(curSources, 0, nCnt*sizeof(int));
        memcpy(curSources, frontier, frontierLength*sizeof(int));
        memset(frontier, 0, nCnt*sizeof(int));
        curSourcesLength = frontierLength;
        colCounter++;
    }
    free(inDegrees);
    free(curSources);
    free(frontier);
    free(frwrd_offsets);
    free(frwrd_csr);
    return colCounter;
}




__global__ void srcKernel(Network ntwk, int cycleID) {
    // Get the ID of the i-th source node
    int i = threadIdx.x;
    int nodeID = ntwk.sourceIDs[i];

    // Set output as the source bit from the right cycle-set
    ntwk.outputBits[cycleID*ntwk.nodeCnt + nodeID] = ntwk.sourceBits[cycleID*ntwk.sourceCnt + i];
}

__global__ void genKernel(Network ntwk, int colNum, int cycleID) {
    // Get the ID of the i-th node in the column
    int i = threadIdx.x;
    int nodeID = ntwk.colIDs[ntwk.colIdxs[colNum] + i];
    int cycleSet = cycleID*ntwk.nodeCnt;

    // Get the IDs of its previous nodes by looking up in offsets and csr
    int prvNdStartIdx = ntwk.offsets[nodeID];
    int width = ntwk.offsets[nodeID+1]-ntwk.offsets[nodeID];                            //<TODO> if this quantity is 1 then is object for sure
                                                                                        //a register? If so can do away with regIndicator.
    int lookupIdx = 0;

    // Get the outputs from the previous nodes (these are the inputs to the i-th node)
    for (int j=0; j<width; j++) {
        int prvNdID = ntwk.csr[j + prvNdStartIdx];
        int prvOutput = ntwk.outputBits[cycleSet + prvNdID];
        if (prvOutput) {
            lookupIdx |= (1 << width-1-j);
        }
    }

    if (ntwk.regIndicator[nodeID]) {
        // Node is a register, send old-gen-val to outputBits and new-gen-val to regBuffer
        int regID = ntwk.typeIDs[nodeID];
        ntwk.outputBits[cycleSet + nodeID] = ntwk.regBuffer[regID];
        ntwk.regBuffer[regID] = lookupIdx;
    } else {
        // Node is a LUT, find the correct lutBit to bitmask with lookupIdx
        int lutID = ntwk.typeIDs[nodeID];
        int intSizeInBits = 8*sizeof(int);
        int quotient = lookupIdx / intSizeInBits;
        int residue = lookupIdx % intSizeInBits;
        int mask = 1 << residue;
        int sectionIdx = ntwk.lutOffsets[lutID] + quotient;
        int sectionBits = ntwk.lutBits[sectionIdx];

        // Get LUT output by bitmasking the residue of lookupIdx against the section from lutBits
        if (mask & sectionBits) {
            ntwk.outputBits[cycleSet + nodeID] = 1;
        } else {
            ntwk.outputBits[cycleSet + nodeID] = 0;
        }
    }
}

















int reader(int* ndCnt, int* edgCnt, int* lutCnt, int* regCnt, int*srcCnt,
        std::vector<int>& regInd_vec,
        std::vector<int>& typeIDs_vec,
        std::vector<int>& srcIDs_vec,
        std::vector<int>& offs_vec,
        std::vector<int>& csr_vec,
        std::vector<int>& lutOffs_vec,
        std::vector<unsigned int>& lutBts_vec) {

    // Define max lengths and container for lines
    int maxLineChars = 1000;
    int maxDigits = 6;
    char line[maxLineChars];

    // Open target file
    FILE *file = fopen("graph.txt", "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening graph.txt\n");
        return 1;
    }

    // Read line 0 for nodecount
    if (fgets(line, maxLineChars, file) != NULL) {
        *ndCnt = atoi(line);
    } else {
        fprintf(stderr, "Error parsing nodecount from line: %s\n", line);
        return 1;
    }

    // Clear line
    memset(line, 0, maxLineChars*sizeof(char));

    // Initialize certain arrays
    std::vector<int> indctr(*ndCnt, 0);
    std::vector<int> typIDs(*ndCnt, 0);
    lutOffs_vec.push_back(0);
    offs_vec.push_back(0);

    // Read file line by line
    int lnCntr = 0;
    int ltOffsCntr = 0;
    int offsCntr = 0;
    while (fgets(line, maxLineChars, file) != NULL) {
        lnCntr++;
        char nodeID[maxDigits], lutWidth[maxDigits], lutInfo[maxLineChars], regDependencies[maxLineChars];
        if (sscanf(line, "%s INPUT %s", nodeID, nodeID) == 2) {                         // Process INPUT
            int id = atoi(nodeID);
            srcIDs_vec.push_back(id);
            typIDs[id] = *srcCnt;
            offs_vec.push_back(offsCntr);
            (*srcCnt)++;
        } else if (sscanf(line, "%s REG %s", nodeID, regDependencies) == 2) {           // Process REG
            int id = atoi(nodeID);
            indctr[id] = 1;
            typIDs[id] = *regCnt;
            offsCntr++;
            offs_vec.push_back(offsCntr);
            std::string dep(regDependencies);
            int depID = std::stoi(dep.substr(1, dep.length()-1));
            csr_vec.push_back(depID);

            (*edgCnt)++;
            (*regCnt)++;
        } else if (sscanf(line, "%s LUT %s %[^\n]", nodeID, lutWidth, lutInfo) == 3) {  // Process LUT
            int id = atoi(nodeID);
            int width = atoi(lutWidth);
            int intSize = 8*sizeof(int);
            int numIntsNeeded = 1;
            if (width > intSize) {
                numIntsNeeded = width / intSize;
                if (width % intSize != 0) {
                    numIntsNeeded++;
                }
            }
            // update lutOffs
            ltOffsCntr += numIntsNeeded;
            lutOffs_vec.push_back(ltOffsCntr);

            // retrieve lut bits and dependencies
            unsigned int curSum = 0;
            int bitCounter = 0;

            // consolidate all lut bits into one number
            int numFlushed = 0;
            for (int j=0; j<width; j++) {
                char curBit;
                curBit = lutInfo[2*j];
                curSum += (atoi(&curBit) << (width-j-1));
                bitCounter++;
                if (bitCounter >= 32) {
                    // flush current int to lutBits and reset curSum and bitCounter
                    lutBts_vec.push_back(curSum);
                    bitCounter = 0;
                    curSum = 0;
                    numFlushed++;
                }
            }

            // flush curSum to lutBits if within numIntsNeeded
            if (numFlushed < numIntsNeeded) {
                lutBts_vec.push_back(curSum);
            }

            // handle dependencies
            std::string info(lutInfo);
            std::string deps = info.substr(2*width, info.length()-1);
            std::string dep;
            std::stringstream strStrm(deps);
            while (std::getline(strStrm, dep, ' ')) {
                // test for validity of dependency
                if (dep.find('D' != std::string::npos)) {
                    // increment edge and offset counts
                    (*edgCnt)++;
                    offsCntr++;
                    std::string depID = dep.substr(1, dep.length()-1);
                    // append to csr
                    csr_vec.push_back(std::stoi(depID));
                }
            }

            // update offsets vector
            offs_vec.push_back(offsCntr);

            typIDs[id] = *lutCnt;
            (*lutCnt)++;
        } else {
            fprintf(stderr, "Error parsing line: %s\n", line);
        }
    }
    regInd_vec = indctr;
    typeIDs_vec = typIDs;
    fclose(file);
    return 0;
}











/*
SPECIFIED BY USER INPUT WAVEFORM (TODO):
    cycleCnt
    srcBts

HANDLED BY COLUMNIZATION:
    colIdxs
    colIDs

LEFT UNSPECIFIED (no action needed):
    regBuf
    outBts
*/




int main() {
    // Int containers to be populated in reader function
    int* numNodes = new int(0);
    int* numEdges = new int(0);
    int* numLuts = new int(0);
    int* numRegs = new int(0);
    int* numSrcs = new int(0);

    // Vectors to be populated in reader function
    std::vector<int> srcIDs_vec;
    std::vector<int> offs_vec;
    std::vector<int> csr_vec;
    std::vector<int> lutOffs_vec;
    std::vector<unsigned int> lutBts_vec;
    std::vector<int> regInd_vec;
    std::vector<int> typeIDs_vec;

    // Call reader function; this fills all num<field> vars and populates vectors
    if (reader(numNodes, numEdges, numLuts, numRegs, numSrcs,
                regInd_vec, typeIDs_vec, srcIDs_vec, offs_vec, csr_vec, lutOffs_vec, lutBts_vec)) {
        return 1;
    }

    // These fields are passed by value to the device
    int cycleCnt = 5;           //<TODO> automate the initialization
                                //of this variable when the user
                                //specifies their input waveform
    int colCnt = 0;
    int ndCnt = *numNodes;
    int edgCnt = *numEdges;
    int lutCnt = *numLuts;
    int regCnt = *numRegs;
    int srcCnt = *numSrcs;

    // These fields are sent to the device and copied as pointers
    // Note that these are host-accessible only
    //<TODO><TODO><TODO><TODO> set all as new int[size] and then later memcpy(vec.data()->arr);
    int* host_srcIDs = new int[srcCnt];
    int* host_srcBts = new int[srcCnt*cycleCnt];
    int* host_offs = new int[1+ndCnt];
    int* host_csr = new int[edgCnt];
    int* host_typeIDs = new int[ndCnt];
    int* host_regInd = new int[ndCnt];
    int* host_regBuf = new int[regCnt];
    int* host_lutOffs = new int[1+lutCnt];
    unsigned int* host_lutBts = new unsigned int[lutCnt];
    int* host_outBts = new int[ndCnt*cycleCnt];
    int* host_colIdxs = new int[1+ndCnt];
    int* host_colIDs =  new int[ndCnt];

    /*    host_srcIDs = new int[srcCnt]{0,1,2,3};
    host_srcBts = new int[srcCnt*cycleCnt]{0,0,0,0, 1,0,0,0, 1,0,0,0, 1,1,1,1, 0,1,1,0};
    host_offs = new int[ndCnt+1]{0,0,0,0,0,2,4,5,7};
    host_csr = new int[edgCnt]{0,1,2,3,4,6,5};
    host_typeIDs = new int[ndCnt]{0,1,2,3,0,1,0,2};
    host_regInd = new int[ndCnt]{0,0,0,0,0,0,1,0};
    host_regBuf = new int[regCnt];
    host_lutOffs = new int[lutCnt+1]{0,1,2,3};
    host_lutBts = new unsigned int[lutCnt]{4,14,1};
    host_outBts = new int[ndCnt*cycleCnt];
    host_colIdxs = new int[ndCnt+1];                                        //<CAREFUL> we're allocating this for ndCnt+1 because we really want colCnt+1
    host_colIDs = new int[ndCnt];                                           // but we calculate colCnt only once we columnize and populate these fields.
    */

    // Transfer vector info to arrays
    memcpy(host_srcIDs, srcIDs_vec.data(), sizeof(int)*srcCnt);
    memcpy(host_offs, offs_vec.data(), sizeof(int)*(1+ndCnt));
    memcpy(host_csr, csr_vec.data(), sizeof(int)*edgCnt);
    memcpy(host_lutOffs, lutOffs_vec.data(), sizeof(int)*(1+lutCnt));
    memcpy(host_lutBts, lutBts_vec.data(), sizeof(int)*lutCnt);
    memcpy(host_regInd, regInd_vec.data(), sizeof(int)*ndCnt);
    memcpy(host_typeIDs, typeIDs_vec.data(), sizeof(int)*ndCnt);
    /*host_srcIDs = srcIDs_vec.data();
    host_offs = offs_vec.data();
    host_csr = csr_vec.data();
    host_lutOffs = lutOffs_vec.data();
    host_lutBts = lutBts_vec.data();
    host_regInd = regInd_vec.data();
    host_typeIDs = typeIDs_vec.data();*/

    // Transfer int containers
    ndCnt = *numNodes;
    edgCnt = *numEdges;
    lutCnt = *numLuts;
    regCnt = *numRegs;
    srcCnt = *numSrcs;

    // Free containers
    free(numNodes);
    free(numEdges);
    free(numLuts);
    free(numRegs);
    free(numSrcs);

    // Initialize input waveform parameters (TODO - automate this)
    cycleCnt = 5;

    /*host_srcBts = new int[srcCnt*cycleCnt];
    // init input bits to all zeroes (TODO - automate this)
    std::fill(host_srcBts, host_srcBts+(srcCnt*cycleCnt), 0);*/
    host_srcBts = new int[srcCnt*cycleCnt]{0,0,0,0, 1,0,0,0, 1,0,0,0, 1,1,1,1, 0,1,1,0};

    host_outBts = new int[ndCnt*cycleCnt];
    host_colIdxs = new int[ndCnt+1];
    host_colIDs = new int[ndCnt];

    // One chief_net to rule them all, one chief_net to find them.
    Network chief_net = Network(srcCnt, host_srcIDs, host_srcBts,
            ndCnt, edgCnt, colCnt, lutCnt, regCnt, host_offs, host_csr,
            host_typeIDs, host_regInd, host_regBuf, host_lutOffs,
            host_lutBts, host_outBts, host_colIdxs, host_colIDs);

    // Columnize and set the number of columns
    chief_net.colCnt = columnize(chief_net);
    colCnt = chief_net.colCnt;

    // These fields exist on the device
    int* dev_srcIDs;
    int* dev_srcBts;
    int* dev_offs;
    int* dev_csr;
    int* dev_typeIDs;
    int* dev_regInd;
    int* dev_regBuf;
    int* dev_lutOffs;
    unsigned int* dev_lutBts;
    int* dev_outBts;
    int* dev_colIdxs;
    int* dev_colIDs;

    // Allocate space for all pointers we're copying
    hipMalloc((void**) &(dev_srcIDs), sizeof(int)*srcCnt);
    hipMalloc((void**) &(dev_srcBts), sizeof(int)*srcCnt*cycleCnt);
    hipMalloc((void**) &(dev_offs), sizeof(int)*(ndCnt+1));
    hipMalloc((void**) &(dev_csr), sizeof(int)*edgCnt);
    hipMalloc((void**) &(dev_typeIDs), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_regInd), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_regBuf), sizeof(int)*regCnt);
    hipMalloc((void**) &(dev_lutOffs), sizeof(int)*(lutCnt+1));
    hipMalloc((void**) &(dev_lutBts), sizeof(int)*lutCnt);
    hipMalloc((void**) &(dev_outBts), sizeof(int)*ndCnt*cycleCnt);
    hipMalloc((void**) &(dev_colIdxs), sizeof(int)*(colCnt+1));
    hipMalloc((void**) &(dev_colIDs), sizeof(int)*ndCnt);

    // Copy contents from host to device
    hipMemcpy(dev_srcIDs, host_srcIDs, sizeof(int)*srcCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_srcBts, host_srcBts, sizeof(int)*srcCnt*cycleCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_offs, host_offs, sizeof(int)*(ndCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_csr, host_csr, sizeof(int)*edgCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_typeIDs, host_typeIDs, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_regInd, host_regInd, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_regBuf, host_regBuf, sizeof(int)*regCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_lutOffs, host_lutOffs, sizeof(int)*(lutCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_lutBts, host_lutBts, sizeof(int)*lutCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_outBts, host_outBts, sizeof(int)*ndCnt*cycleCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_colIdxs, host_colIdxs, sizeof(int)*(colCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_colIDs, host_colIDs, sizeof(int)*ndCnt, hipMemcpyHostToDevice);

    // Point to device pointers from chief_net
    chief_net.sourceIDs = dev_srcIDs;
    chief_net.sourceBits = dev_srcBts;
    chief_net.offsets = dev_offs;
    chief_net.csr = dev_csr;
    chief_net.typeIDs = dev_typeIDs;
    chief_net.regIndicator = dev_regInd;
    chief_net.regBuffer = dev_regBuf;
    chief_net.lutOffsets = dev_lutOffs;
    chief_net.lutBits = dev_lutBts;
    chief_net.outputBits = dev_outBts;
    chief_net.colIdxs = dev_colIdxs;
    chief_net.colIDs = dev_colIDs;

    // Run cycle simulation on the kernels                          <TODO> possible optimization: try merging the kernels and calling one kernel on colCnt many synchronized blocks
                                                                        // instead, since this will minimize CPU-GPU communication latency and make the bulk of the simulation internal on the GPU.
    for (int k=0; k<cycleCnt; k++) {
        printf("\n<<<<<<<<<entering cycle [%d]>>>>>>>>", k);
        // Invoke srcKernel once using srcCnt many threads
        srcKernel<<<1, srcCnt>>>(chief_net, k);
        hipDeviceSynchronize();

        // Invoke genKernel once per non-source column (in-order), using as many threads as there are nodes in the column
        for (int i=1; i<colCnt; i++) {
            int columnHeight = host_colIdxs[i+1]-host_colIdxs[i];
            genKernel<<<1, columnHeight>>>(chief_net, i, k);
            hipDeviceSynchronize();
        }
    }

    // Copy results from device to host
    hipMemcpy(host_outBts, dev_outBts, sizeof(int)*ndCnt*cycleCnt, hipMemcpyDeviceToHost);

    // Print simulation outputs
    printf("\n");
    for (int k=0; k<cycleCnt; k++) {
        printf(" _______________________________\n");
        printf("|        CYCLE[%d] RESULTS       |\n", k);
        printf("|-------------------------------|\n");
        for (int j=0; j<ndCnt; j++) {
            printf("|    Node[%d]'s output is [%d]    |\n", j, host_outBts[k*ndCnt + j]);
        }
        printf("|_______________________________|\n");
    }

    // Free device pointers
    hipFree(dev_srcIDs);
    hipFree(dev_srcBts);
    hipFree(dev_offs);
    hipFree(dev_csr);
    hipFree(dev_typeIDs);
    hipFree(dev_regInd);
    hipFree(dev_regBuf);
    hipFree(dev_lutOffs);
    hipFree(dev_lutBts);
    hipFree(dev_outBts);
    hipFree(dev_colIdxs);
    hipFree(dev_colIDs);

    // Free host pointers
    free(host_srcIDs);
    free(host_srcBts);
    free(host_offs);
    free(host_csr);
    free(host_typeIDs);
    free(host_regInd);
    free(host_regBuf);
    free(host_lutOffs);
    free(host_lutBts);
    free(host_outBts);
    free(host_colIdxs);
    free(host_colIDs);

    return 0;
}








