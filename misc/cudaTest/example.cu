
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    // 2 is number of computational blocks and (size + 1) / 2 is a number of threads in a block
    addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    const int arraySize = 5;
    const int a[arraySize] = {  1,  2,  3,  4,  5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    addWithCuda(c, a, b, arraySize);
    printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);
    hipDeviceReset();
    return 0;
}



