
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <vector>





struct Network {
    // Handled by srcKernel
    int sourceCnt;              // number of source nodes
    int* sourceIDs;             // sourceIDs[i] = ID of the i-th source node
    int* sourceBits;            // decimal-formatted source vals for k-th cycle are found through indices: k*sourceCnt -> (k+1)*sourceCnt-1

    // Handled by genKernel
    int nodeCnt;                // number of nodes
    int edgeCnt;                // number of edges
    int colCnt;                 // number of columns
    int lutCnt;                 // number of LUTs
    int regCnt;                 // number of registers
    int* offsets;               // offsets[i] = prefix sum of #prevNodes from node 0 to i-1, inclusive 
    int* csr;                   // i-th node's prevNode IDs are found:     csr[offs[i]] -> csr[offs[i+1]-1]
    int* typeIDs;               // typeIDs[i] = n where node with ID i is the n-th input, LUT, or register, 0-indexed (to get INP/LUT/REG-ID given nodeID)
    int* regIndicator;          // regIndicator[i] = 1 if node @ ID i is a register, 0 otherwise.
    int* regBuffer;             // regBuffer[i] = regID i register's oldGen val before being touched by genKernel, newGen val after being touched. 
    int* lutOffsets;            // lutOffs[i] = prefix sum of #ints-needed-to-represent-LUT-control-bits from LUT 0 to i-1, inclusive (offsets for the lutBits-CSR).
    unsigned int* lutBits;      // i-th LUT's lutbits (decimal-formatted) are found:         lutBits[lutOffsets[i]] -> lutBits[lutOffsets[i+1]-1]
    int* outputBits;            // decimal-formatted output vals for k-th cycle are found through indices: k*nodeCnt -> (k+1)*nodeCnt-1
    int* colIdxs;               // colIdxs[i] = prefix sum of column-nodecounts from col 0 to i-1, inclusive 
    int* colIDs;                // i-th column's nodeIDs are found:        colIDs[colIdxs[i]] -> colIDs[colIdxs[i+1]-1]

    Network(int srcCnt, int* srcIDs, int* srcBts, int ndCnt, int edgCnt, int colCnt, int lutCnt, int regCnt, int* offs, int* csr,
            int* tpIDs, int* regInd, int* regBuf, int* lutOffs, unsigned int* lutBts, int* outBts, int* colIdxs, int* colIDs)
       : sourceCnt(srcCnt),
         sourceIDs(srcIDs),
         sourceBits(srcBts),
         nodeCnt(ndCnt),
         edgeCnt(edgCnt),
         colCnt(colCnt),
         lutCnt(lutCnt),
         regCnt(regCnt),
         offsets(offs),
         csr(csr),
         typeIDs(tpIDs),
         regIndicator(regInd),
         regBuffer(regBuf),
         lutOffsets(lutOffs),
         lutBits(lutBts),
         outputBits(outBts),
         colIdxs(colIdxs),
         colIDs(colIDs) {} 
};







int comparator(const void *a, const void *b) {
    return ((const int (*)[2])a)[0][0] - ((const int (*)[2])b)[0][0];
}

// This csr-reversal function WILL NOT preserve the order of dependencies (matters to LUT computation)
void gen_outgoing_csr(Network ntwk, int* out_offs, int* out_csr) {
    // Form tuples of all (srcID, dstID) connexions from the graph
    int tupleCnt = 0;
    int tuples[ntwk.edgeCnt][2];         // k-th tuple located at: (tuples[k][0], tuples[k][1]])
    int* outDegrees = new int[ntwk.nodeCnt];        // outDegrees[nodeID] = out-degree of corresponding node

    for (int dstID=0; dstID<=ntwk.nodeCnt; dstID++) {
        int numDependencies = ntwk.offsets[dstID+1]-ntwk.offsets[dstID];
        for (int i=0; i<numDependencies; i++) {
            int srcID = ntwk.csr[ntwk.offsets[dstID]+i];
            tuples[tupleCnt][0] = srcID;
            tuples[tupleCnt][1] = dstID;
            outDegrees[srcID]++;
            tupleCnt++;
        }
    }

    // Sort tuples by srcID, in ascending order
    qsort((void*)tuples, ntwk.edgeCnt, sizeof(int*), comparator);

    // Configure offsets for outgoing-csr
    out_offs[0] = 0;
    int appendCntr = 0;
    for (int source=0; source<ntwk.nodeCnt; source++) {
        out_offs[source+1] = out_offs[source] + outDegrees[source];
        for (int i=0; i<outDegrees[source]; i++) {
            out_csr[out_offs[source]+i] = tuples[appendCntr][1];
            appendCntr++;
        }
    }

    free(outDegrees);
}

// Columnizes a network. Assumes that all fields are set except for colIdxs and colIDs
int columnize(Network ntwk) {
    int colCounter = 0;
    int nCnt = ntwk.nodeCnt;
    int sCnt = ntwk.sourceCnt;
    int curSourcesLength = sCnt;
    int* inDegrees = new int[nCnt];
    int* curSources = new int[nCnt];
    int* frontier = new int[nCnt];
    int* frwrd_offsets = new int[ntwk.nodeCnt+1];
    int* frwrd_csr = new int[ntwk.edgeCnt];

    // Generate forward-edge (outgoing) graph representation
    gen_outgoing_csr(ntwk, frwrd_offsets, frwrd_csr);

    // Populate inDegrees
    for (int k=0; k<nCnt; k++) {
        inDegrees[k] = ntwk.offsets[k+1]-ntwk.offsets[k];
    }

    // Populate curSources with source IDs to begin with
    memcpy(curSources, ntwk.sourceIDs, sCnt*sizeof(int));
    int curColIDsLength = 0;
    
    // Loop until all nodes have been placed into a column
    while (curSourcesLength != 0) {
        // Initialize next entry in colIdxs to continue the prefix sum
        ntwk.colIdxs[colCounter+1] = ntwk.colIdxs[colCounter];
        int frontierLength = 0;
        
        for (int i=0; i<curSourcesLength; i++) {
            // Flush each curSource to a column and decrement its inDegree so it isn't picked up again
            int nodeID = curSources[i];
            ntwk.colIDs[curColIDsLength] = nodeID;
            curColIDsLength++;
            inDegrees[nodeID]--;

            // Bump the current column's offset
            ntwk.colIdxs[colCounter+1]++;

            // Decrement inDegree count for each node outgoing from a curSource
            int outDeg = frwrd_offsets[nodeID+1]-frwrd_offsets[nodeID];
            for (int k=0; k<outDeg; k++) {
                inDegrees[frwrd_csr[frwrd_offsets[nodeID]+k]]--;
            }
        }

        // Append all nodes that have exactly 0 inDegree to the frontier
        for (int j=0; j<nCnt; j++) {
            if (inDegrees[j] == 0) {
                frontier[frontierLength] = j;
                frontierLength++;
            }
        }
   
        // Flush frontier to curSources
        // I think this can all be optimized by actually dual-buffering here. Too much memcpy...
        memset(curSources, 0, nCnt*sizeof(int));
        memcpy(curSources, frontier, frontierLength*sizeof(int));
        memset(frontier, 0, nCnt*sizeof(int));
        curSourcesLength = frontierLength;
        colCounter++;
    }
    free(inDegrees);
    free(curSources);
    free(frontier);
    free(frwrd_offsets);
    free(frwrd_csr);
    return colCounter;
}







__global__ void srcKernel(Network ntwk, int cycleID) {
    // Get the ID of the i-th source node
    int i = threadIdx.x;
    int nodeID = ntwk.sourceIDs[i];

    // Set output as the source bit from the right cycle-set
    ntwk.outputBits[cycleID*ntwk.nodeCnt + nodeID] = ntwk.sourceBits[cycleID*ntwk.sourceCnt + i];
}

__global__ void genKernel(Network ntwk, int colNum, int cycleID) {
    // Get the ID of the i-th node in the column
    int i = threadIdx.x;
    int nodeID = ntwk.colIDs[ntwk.colIdxs[colNum] + i];
    int cycleSet = cycleID*ntwk.nodeCnt;

    // Get the IDs of its previous nodes by looking up in offsets and csr
    int prvNdStartIdx = ntwk.offsets[nodeID];
    int width = ntwk.offsets[nodeID+1]-ntwk.offsets[nodeID];                            //<TODO> if this quantity is 1 then is object for sure a register? If so can do away with regIndicator.
    int lookupIdx = 0;

    // Get the outputs from the previous nodes (these are the inputs to the i-th node)
    for (int j=0; j<width; j++) {
        int prvNdID = ntwk.csr[j + prvNdStartIdx];
        int prvOutput = ntwk.outputBits[cycleSet + prvNdID];
        if (prvOutput) {
            lookupIdx |= (1 << width-1-j);
        }
    }

    if (ntwk.regIndicator[nodeID]) {
        // Node is a register, send old-gen-val to outputBits and new-gen-val to regBuffer
        int regID = ntwk.typeIDs[nodeID];
        ntwk.outputBits[cycleSet + nodeID] = ntwk.regBuffer[regID]; 
        ntwk.regBuffer[regID] = lookupIdx;
    } else {
        // Node is a LUT, find the correct lutBit to bitmask with lookupIdx
        int lutID = ntwk.typeIDs[nodeID];
        int intSizeInBits = 8*sizeof(int);
        int quotient = lookupIdx / intSizeInBits;
        int residue = lookupIdx % intSizeInBits;
        int mask = 1 << residue;
        int sectionIdx = ntwk.lutOffsets[lutID] + quotient;
        int sectionBits = ntwk.lutBits[sectionIdx];

        // Get LUT output by bitmasking the residue of lookupIdx against the section from lutBits
        if (mask & sectionBits) {
            ntwk.outputBits[cycleSet + nodeID] = 1;
        } else {
            ntwk.outputBits[cycleSet + nodeID] = 0;
        }
    }
}
















int main() {
    std::vector<int> jej;
    jej.push_back(6);









    int cycleCnt;
    int ndCnt;
    int edgCnt;
    int colCnt;
    int lutCnt;
    int regCnt;
    int srcCnt;
    
    // These fields are sent to the device and copied as pointers
    // Note that these are host-accessible only
    int* host_srcIDs;
    int* host_srcBts;
    int* host_offs;
    int* host_csr;
    int* host_typeIDs;
    int* host_regInd;
    int* host_regBuf;
    int* host_lutOffs;
    unsigned int* host_lutBts;
    int* host_outBts;
    int* host_colIdxs;
    int* host_colIDs;

    //=====================================================================//
    //===========================5-cycle sim test==========================//
    //=====================================================================//
    cycleCnt = 5;

    ndCnt = 8;
    edgCnt = 7;
    colCnt = 4;
    lutCnt = 3;
    regCnt = 1;
    srcCnt = 4;

    host_srcIDs = new int[srcCnt]{0,1,2,3};
    host_srcBts = new int[srcCnt*cycleCnt]{0,0,0,0, 1,0,0,0, 1,0,0,0, 1,1,1,1, 0,1,1,0};
    host_offs = new int[ndCnt+1]{0,0,0,0,0,2,4,5,7};
    host_csr = new int[edgCnt]{0,1,2,3,4,6,5};
    host_typeIDs = new int[ndCnt]{0,1,2,3,0,1,0,2};
    host_regInd = new int[ndCnt]{0,0,0,0,0,0,1,0};
    host_regBuf = new int[regCnt];
    host_lutOffs = new int[lutCnt+1]{0,1,2,3};
    host_lutBts = new unsigned int[lutCnt]{4,14,1};
    host_outBts = new int[ndCnt*cycleCnt];
    host_colIdxs = new int[ndCnt+1];                                        //<CAREFUL> we're allocating this for ndCnt+1 because we really want colCnt+1
    host_colIDs = new int[ndCnt];                                           // but we calculate colCnt only once we columnize and populate these fields.
                                                                            // Is it fine only sending colCnt+1 many ints to/from host/device with cudaMemcpys?
    // One chief_net to rule them all, one chief_net to find them.
    Network chief_net = Network(srcCnt, host_srcIDs, host_srcBts, ndCnt, edgCnt, colCnt, lutCnt, regCnt,
            host_offs, host_csr, host_typeIDs, host_regInd, host_regBuf, host_lutOffs, host_lutBts, host_outBts, host_colIdxs, host_colIDs);

    chief_net.colCnt = columnize(chief_net);
    //=====================================================================//    if the above is a problem, then you *could* create another colIdxs of the right size,
    //=====================================================================//    then memcpy and make chief_net point to *that*, and then run free() on the old one.
    //=====================================================================//    but probably not worth the effort, let's save this for later unless things fall apart.

    // These fields exist on the device
    int* dev_srcIDs;
    int* dev_srcBts;
    int* dev_offs;
    int* dev_csr;
    int* dev_typeIDs;
    int* dev_regInd;
    int* dev_regBuf;
    int* dev_lutOffs;
    unsigned int* dev_lutBts;
    int* dev_outBts;
    int* dev_colIdxs;
    int* dev_colIDs;

    // Allocate space for all pointers we're copying 
    hipMalloc((void**) &(dev_srcIDs), sizeof(int)*srcCnt);
    hipMalloc((void**) &(dev_srcBts), sizeof(int)*srcCnt*cycleCnt);
    hipMalloc((void**) &(dev_offs), sizeof(int)*(ndCnt+1));
    hipMalloc((void**) &(dev_csr), sizeof(int)*edgCnt);
    hipMalloc((void**) &(dev_typeIDs), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_regInd), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_regBuf), sizeof(int)*regCnt);
    hipMalloc((void**) &(dev_lutOffs), sizeof(int)*(lutCnt+1));
    hipMalloc((void**) &(dev_lutBts), sizeof(int)*lutCnt);
    hipMalloc((void**) &(dev_outBts), sizeof(int)*ndCnt*cycleCnt);
    hipMalloc((void**) &(dev_colIdxs), sizeof(int)*(colCnt+1));
    hipMalloc((void**) &(dev_colIDs), sizeof(int)*ndCnt);

    // Copy contents from host to device
    hipMemcpy(dev_srcIDs, host_srcIDs, sizeof(int)*srcCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_srcBts, host_srcBts, sizeof(int)*srcCnt*cycleCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_offs, host_offs, sizeof(int)*(ndCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_csr, host_csr, sizeof(int)*edgCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_typeIDs, host_typeIDs, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_regInd, host_regInd, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_regBuf, host_regBuf, sizeof(int)*regCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_lutOffs, host_lutOffs, sizeof(int)*(lutCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_lutBts, host_lutBts, sizeof(int)*lutCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_outBts, host_outBts, sizeof(int)*ndCnt*cycleCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_colIdxs, host_colIdxs, sizeof(int)*(colCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_colIDs, host_colIDs, sizeof(int)*ndCnt, hipMemcpyHostToDevice);

    // Point to device pointers from chief_net
    chief_net.sourceIDs = dev_srcIDs;
    chief_net.sourceBits = dev_srcBts;
    chief_net.offsets = dev_offs;
    chief_net.csr = dev_csr;
    chief_net.typeIDs = dev_typeIDs;
    chief_net.regIndicator = dev_regInd;
    chief_net.regBuffer = dev_regBuf;
    chief_net.lutOffsets = dev_lutOffs;
    chief_net.lutBits = dev_lutBts;
    chief_net.outputBits = dev_outBts;
    chief_net.colIdxs = dev_colIdxs;
    chief_net.colIDs = dev_colIDs;

    // Run cycle simulation on the kernels                          <TODO> possible optimization: try merging the kernels and calling one kernel on colCnt many synchronized blocks
                                                                        // instead, since this will minimize CPU-GPU communication latency and make the bulk of the simulation internal on the GPU.
    for (int k=0; k<cycleCnt; k++) {
        printf("\n<<<<<<<<<entering cycle [%d]>>>>>>>>", k); 
        // Invoke srcKernel once using srcCnt many threads
        srcKernel<<<1, srcCnt>>>(chief_net, k);
        hipDeviceSynchronize();

        // Invoke genKernel once per non-source column (in-order), using as many threads as there are nodes in the column
        for (int i=1; i<colCnt; i++) {
            int columnHeight = host_colIdxs[i+1]-host_colIdxs[i];
            genKernel<<<1, columnHeight>>>(chief_net, i, k);
            hipDeviceSynchronize();
        }
    }

    // Copy results from device to host
    hipMemcpy(host_outBts, dev_outBts, sizeof(int)*ndCnt*cycleCnt, hipMemcpyDeviceToHost);

    // Print simulation outputs
    printf("\n");
    for (int k=0; k<cycleCnt; k++) {
        printf(" _______________________________\n");
        printf("|        CYCLE[%d] RESULTS       |\n", k); 
        printf("|-------------------------------|\n");
        for (int j=0; j<ndCnt; j++) {
            printf("|    Node[%d]'s output is [%d]    |\n", j, host_outBts[k*ndCnt + j]);
        }
        printf("|_______________________________|\n");
    }

    // Free device pointers
    hipFree(dev_srcIDs);
    hipFree(dev_srcBts);
    hipFree(dev_offs);
    hipFree(dev_csr);
    hipFree(dev_typeIDs);
    hipFree(dev_regInd);
    hipFree(dev_regBuf);
    hipFree(dev_lutOffs);
    hipFree(dev_lutBts);
    hipFree(dev_outBts);
    hipFree(dev_colIdxs);
    hipFree(dev_colIDs);

    // Free host pointers
    free(host_srcIDs);
    free(host_srcBts);
    free(host_offs);
    free(host_csr);
    free(host_typeIDs);
    free(host_regInd);
    free(host_regBuf);
    free(host_lutOffs);
    free(host_lutBts);
    free(host_outBts);
    free(host_colIdxs);
    free(host_colIDs);

    return 0;
}









