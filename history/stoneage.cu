
#include <hip/hip_runtime.h>
#include <stdio.h>





struct Network {
    // Handled by srcKernel
    int sourceCnt;              // number of source nodes
    int* sourceIDs;             // sourceIDs[i] = ID of the i-th source node
    int* sourceBits;            // sourceBits[i] = i-th source node's integer-formatted input bitvalue 

    // Handled by genKernel
    int nodeCnt;                // number of nodes
    int edgeCnt;                // number of edges
    int colCnt;                 // number of columns
    int* offsets;               // i-th node's prevNode IDs are found:     csr[offs[i]] -> csr[offs[i+1]-1]
    int* csr;                   // contains prevNode IDs as shown above
    int lutWidth;               // standard bitwidth of a LUT (assuming constant)
    int* lutBits;               // lutBits[i] = i-th node's lutbits, integer-formatted bitvalues
    int* outputBits;            // outputBits[i] = i-th node's integer-formatted output bitvalue
    int* colIdxs;               // colIdxs[i] = prefix sum of column nodecounts from col 0 to i-1, inclusive (so colIdxs[0]=0, colIdxs[1]=#ndsCol0, colIdxs[n]=sum(#ndsCol0->n-1)) 
    int* colIDs;                // i-th column's nodeIDs are found:        colIDs[colIdxs[i]] -> colIDs[colIdxs[i+1]-1]

    Network(int srcCnt,int* srcIDs,int* srcBts,int ndCnt,int edgCnt,int colCnt,int* offs,int* csr,int lutWdth,int* lutBts,int* outBts,int* colIdxs,int* colIDs)
       : sourceCnt(srcCnt),
         sourceIDs(srcIDs),
         sourceBits(srcBts),
         nodeCnt(ndCnt),
         edgeCnt(edgCnt),
         colCnt(colCnt),
         offsets(offs),
         csr(csr),
         lutWidth(lutWdth),
         lutBits(lutBts),
         outputBits(outBts),
         colIdxs(colIdxs),
         colIDs(colIDs) {} 
};





__global__ void srcKernel(Network ntwk) {
    // Looking at the i-th node in the source column
    int i = threadIdx.x;

    // Get the ID of the i-th source node
    int nodeID = ntwk.sourceIDs[i];
    printf("Thread [%d], NodeID [%d]\n", i, nodeID);

    // Set output by bitmasking sourceBits against lutBits
    int mask = 1 << ntwk.sourceBits[i];

    if (mask & ntwk.lutBits[nodeID]) {
        ntwk.outputBits[nodeID] = 1;
    } else {
        ntwk.outputBits[nodeID] = 0;
    }
}




__global__ void genKernel(Network ntwk, int colNum) {
    // Looking at the i-th node in the column
    int i = threadIdx.x;

    // Get the ID of the i-th node in the column
    int nodeID = ntwk.colIDs[ntwk.colIdxs[colNum] + i];

    // Get the IDs of his previous nodes by looking up in offsets and csr
    int prvNdStartIdx = ntwk.offsets[nodeID];
    int width = ntwk.lutWidth;
    int lutLookupIdx = 0;

    // Get the outputs from the previous nodes (these are the inputs to the i-th node)
    for (int j=0; j<width; j++) {
        int prvNdID = ntwk.csr[j + prvNdStartIdx];
        int prvOutput = ntwk.outputBits[prvNdID];
        if (prvOutput) {
            lutLookupIdx |= (1 << width-1-j);
        }
    }

    // Get lut output by bitmasking lutLookupIdx against lutBits
    int mask = 1 << lutLookupIdx;

    if (mask & ntwk.lutBits[nodeID]) {
        ntwk.outputBits[nodeID] = 1;
    } else {
        ntwk.outputBits[nodeID] = 0;
    }

    printf("Column [%d], Thread [%d], NodeID [%d], LutLookupIdx [%d]\n", colNum, i, nodeID, lutLookupIdx);
}





int main() {
    // These fields are sent to the device, but not copied as pointers
    int ndCnt;
    int edgCnt;
    int colCnt;
    int lutWdth;
    int srcCnt;
    
    // These fields are sent to the device and copied as pointers
    // Note that these are host-accessible only
    int* host_srcIDs;
    int* host_srcBts;
    int* host_offs;
    int* host_csr;
    int* host_lutBts;
    int* host_outBts;
    int* host_colIdxs;
    int* host_colIDs;


    //=====================================================================//
    //============================Test Suite 1=============================//
    //=====================================================================//
    ndCnt = 6;
    edgCnt = 6;
    colCnt = 3;
    lutWdth = 2;
    srcCnt = 3;

    host_srcIDs = new int[srcCnt]{0,1,2};
    host_srcBts = new int[srcCnt]{0,2,0};
    host_offs = new int[ndCnt+1]{0,0,0,0,2,4,6};
    host_csr = new int[edgCnt]{0,1,1,2,3,4};
    host_lutBts = new int[ndCnt]{2,4,8,2,4,8};
    host_outBts = new int[ndCnt];
    host_colIdxs = new int[ndCnt+1]{0,3,5,6};
    host_colIDs = new int[ndCnt]{0,1,2,3,4,5};
    //=====================================================================//
    //=====================================================================//
    //=====================================================================//


    // These fields exist on the device
    int* dev_srcIDs;
    int* dev_srcBts;
    int* dev_offs;
    int* dev_csr;
    int* dev_lutBts;
    int* dev_outBts;
    int* dev_colIdxs;
    int* dev_colIDs;

    // Create a single chief_net to hold all fields
    Network chief_net = Network(srcCnt, host_srcIDs, host_srcBts, ndCnt, edgCnt, colCnt, host_offs, host_csr, lutWdth, host_lutBts, host_outBts, host_colIdxs, host_colIDs);
    
    // Now allocate space for all pointers which must be copied
    hipMalloc((void**) &(dev_srcIDs), sizeof(int)*srcCnt);
    hipMalloc((void**) &(dev_srcBts), sizeof(int)*srcCnt);
    hipMalloc((void**) &(dev_offs), sizeof(int)*(ndCnt+1));
    hipMalloc((void**) &(dev_csr), sizeof(int)*edgCnt);
    hipMalloc((void**) &(dev_lutBts), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_outBts), sizeof(int)*ndCnt);
    hipMalloc((void**) &(dev_colIdxs), sizeof(int)*(colCnt+1));
    hipMalloc((void**) &(dev_colIDs), sizeof(int)*ndCnt);

    // Now copy contents from host to device
    hipMemcpy(dev_srcIDs, host_srcIDs, sizeof(int)*srcCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_srcBts, host_srcBts, sizeof(int)*srcCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_offs, host_offs, sizeof(int)*(ndCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_csr, host_csr, sizeof(int)*edgCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_lutBts, host_lutBts, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_outBts, host_outBts, sizeof(int)*ndCnt, hipMemcpyHostToDevice);
    hipMemcpy(dev_colIdxs, host_colIdxs, sizeof(int)*(colCnt+1), hipMemcpyHostToDevice);
    hipMemcpy(dev_colIDs, host_colIDs, sizeof(int)*ndCnt, hipMemcpyHostToDevice);

    // Point to device pointers from chief_net
    chief_net.sourceIDs = dev_srcIDs;
    chief_net.sourceBits = dev_srcBts;
    chief_net.offsets = dev_offs;
    chief_net.csr = dev_csr;
    chief_net.lutBits = dev_lutBts;
    chief_net.outputBits = dev_outBts;
    chief_net.colIdxs = dev_colIdxs;
    chief_net.colIDs = dev_colIDs;

 
    // Invoke srcKernel once using srcCnt many threads
    srcKernel<<<1, srcCnt>>>(chief_net);

    // Invoke genKernel once per non-source column (in-order), using as many threads as there are nodes in the column
    for (int i=1; i<colCnt; i++) {
        int columnHeight = host_colIdxs[i+1]-host_colIdxs[i];
        genKernel<<<1, columnHeight>>>(chief_net, i);
        // Force sequential col-by-col execution.
        hipDeviceSynchronize();
    }

    // Now copy what's changed from device to host
    hipMemcpy(host_outBts, dev_outBts, sizeof(int)*ndCnt, hipMemcpyDeviceToHost);

    // Point once again to host pointers from chief_net
    chief_net.sourceIDs = host_srcIDs;
    chief_net.sourceBits = host_srcBts;
    chief_net.offsets = host_offs;
    chief_net.csr = host_csr;
    chief_net.lutBits = host_lutBts;
    chief_net.outputBits = host_outBts;
    chief_net.colIdxs = host_colIdxs;
    chief_net.colIDs = host_colIDs;

    // Free the device pointers no longer in use
    hipFree(dev_srcIDs);
    hipFree(dev_srcBts);
    hipFree(dev_offs);
    hipFree(dev_csr);
    hipFree(dev_lutBts);
    hipFree(dev_outBts);
    hipFree(dev_colIdxs);
    hipFree(dev_colIDs);

    // Now print simulation outputs or whatever else is desired
    printf("\n\n");
    for (int k=0; k<ndCnt; k++) {
        printf("Node [%d]'s output value is [%d]\n", k, chief_net.outputBits[k]);
    }

    // Finally, free the host pointers once everything is done
    free(host_srcIDs);
    free(host_srcBts);
    free(host_offs);
    free(host_csr);
    free(host_lutBts);
    free(host_outBts);
    free(host_colIdxs);
    free(host_colIDs);

    return 0;
}


